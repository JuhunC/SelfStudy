﻿#include<iostream>
#include<hip/hip_runtime.h>


__global__ void add(int a, int b, int* c)
{
	*c = a + b;
}

int main(void)
{
	int c;
	int* dev_c;

	hipMalloc((void**)&dev_c, sizeof(int));

	add <<<1, 1 >>> (2, 7, dev_c);
	
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

	printf("%d + %d = %d\n", 2, 7, c);

	hipFree(dev_c);





	return 0;
}