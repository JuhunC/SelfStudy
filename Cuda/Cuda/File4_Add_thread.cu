﻿#include<iostream>
#include<hip/hip_runtime.h>

#define N (1024)

__global__ void add(int* a, int* b, int* c)
{
	int tid = threadIdx.x;
	if (tid < N)
	{
		c[tid] = a[tid] + b[tid];
	}
	return;
}

int main(void) 
{
	int a[N], b[N], c[N];
	int* dev_a, * dev_b, * dev_c;

	hipMalloc((void**)&dev_a, sizeof(int) * N);
	hipMalloc((void**)&dev_b, sizeof(int) * N);
	hipMalloc((void**)&dev_c, sizeof(int) * N);

	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = i * i;
	}

	hipMemcpy(dev_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, sizeof(int) * N, hipMemcpyHostToDevice);

	add <<<1, N >>> (dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

}