#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<>
#include<common/book.h>
#include<chrono>
#include<stdio.h>

#define CUDA_MAX_THREAD	(1024)
#define N				(33 * CUDA_MAX_THREAD)

auto getTime(void)
{
	return std::chrono::high_resolution_clock::now();
}

__global__ void cudaAdd(int* a, int* b, int* c)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	while (tid < N)
	{
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
}
__host__ void Add(int* a, int* b, int* c)
{
	for (int i = 0; i < N; i++)
	{
		c[i] = a[i] + b[i];
	}
}
void CPU(int* a, int* b, int* c)
{
	Add(a, b, c);
}
void GPU(int* a, int* b, int* c)
{
	int* dev_a, * dev_b, * dev_c;

	hipMalloc((void**)&dev_a, sizeof(int) * N);
	hipMalloc((void**)&dev_b, sizeof(int) * N);
	hipMalloc((void**)&dev_c, sizeof(int) * N);

	hipMemcpy(dev_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, sizeof(int) * N, hipMemcpyHostToDevice);
	auto s1 = getTime();

	cudaAdd << <128, 128 >> > (dev_a, dev_b, dev_c);
	auto e1 = getTime();
	auto elapsed = (std::chrono::nanoseconds)(e1 - s1);
	printf("[%.5f seconds] GPU Add Function Runtime\n", elapsed.count() * 1e-9);

	hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	return;
}

int main(void)
{
	int a[N], b[N], c[N];

	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = i * i;
	}
	auto s1 = getTime();
	GPU(a, b, c);
	auto e1 = getTime();
	auto elapsed = (std::chrono::nanoseconds)(e1 - s1);

	printf("[%.5f seconds] GPU Runtime(including MemCpy/Alloc/Dealloc)\n", elapsed.count() * 1e-9);

	auto s2 = getTime();
	CPU(a, b, c);
	auto e2 = getTime();
	auto elapsed1 = (std::chrono::nanoseconds)(e2 - s2);

	printf("[%.5f seconds] CPU Runtime\n", elapsed1.count() * 1e-9);

	return;
}